#ifdef __clang__
hipError_t hipConfigureCall(dim3, dim3, size_t=0, hipStream_t=0);
#endif

// #include <cuda_runtime.h>

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hello_from_gpu()
{
    int x = threadIdx.x;
    printf("Hello there, CUDA from thread %d!\n", x);
}

int main()
{
    hello_from_gpu<<<1,10>>>();
    hipDeviceReset();
    return 0;
}
