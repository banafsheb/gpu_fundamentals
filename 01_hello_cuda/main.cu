#ifdef __clang__
hipError_t hipConfigureCall(dim3, dim3, size_t=0, hipStream_t=0);
#endif


#include <hip/hip_runtime.h>
#include <stdio.h>


// kernel definition
__global__ void hello_from_gpu()
{
    // thread coordinates: blockIdx and threadIdx
    int x = threadIdx.x;
    printf("Hello there, CUDA from thread %d!\n", x);
}

int main()
{
    // kernel invocation
    hello_from_gpu<<<1,10>>>();
    hipDeviceReset();
    return 0;
}
